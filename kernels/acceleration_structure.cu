
#include <hip/hip_runtime.h>
#include <stdint.h>
// #include <optix.h>
// #include <optix_device.h>
// #include <optix_host.h>
// #include <optix_stubs.h>

extern "C" void createAccelerationStructure(void* vertexBuffer, uint32_t vertexCount, void* indexBuffer, uint32_t indexCount) {
  // cudaFree(0);
  // optixInit();

  // OptixDeviceContextOptions options = {};
  // CUcontext cuCtx = 0;
  // OptixDeviceContext context = NULL;
  // optixDeviceContextCreate(cuCtx, &options, &context);

  // OptixAccelBuildOptions accelBuildOptions = {
  //   .buildFlags = OPTIX_BUILD_FLAG_NONE,
  //   .operation = OPTIX_BUILD_OPERATION_BUILD,
  //   .motionOptions = {
  //     .numKeys = 0,
  //     .flags = 0,
  //     .timeBegin = 0,
  //     .timeEnd = 0
  //   }
  // };

  // const uint32_t buildInputTriangleArrayFlags[1] = { OPTIX_GEOMETRY_FLAG_NONE };

  // CUdeviceptr vertexBufferDevice = 0;
  // cudaMalloc((void**)(&vertexBufferDevice), vertexCount * sizeof(float));
  // cudaMemcpy((void*)vertexBufferDevice, vertexBuffer, vertexCount * sizeof(float), cudaMemcpyHostToDevice);

  // CUdeviceptr indexBufferDevice = 0;
  // cudaMalloc((void**)(&indexBufferDevice), indexCount * sizeof(uint32_t));
  // cudaMemcpy((void*)indexBufferDevice, indexBuffer, indexCount * sizeof(uint32_t), cudaMemcpyHostToDevice);

  // OptixBuildInputTriangleArray buildInputTriangleArray = {
  //   .vertexBuffers = &vertexBufferDevice,
  //   .numVertices = vertexCount,
  //   .vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3,
  //   .vertexStrideInBytes = sizeof(float3),
  //   .indexBuffer = indexBufferDevice,
  //   .numIndexTriplets = indexCount / 3,
  //   .indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3,
  //   .indexStrideInBytes = sizeof(int3),
  //   .preTransform = 0,
  //   .flags = buildInputTriangleArrayFlags,
  //   .numSbtRecords = 1,
  //   .sbtIndexOffsetBuffer = 0,
  //   .sbtIndexOffsetSizeInBytes = sizeof(int),
  //   .sbtIndexOffsetStrideInBytes = sizeof(int),
  //   .primitiveIndexOffset = 0,
  //   .transformFormat = OPTIX_TRANSFORM_FORMAT_NONE
  // };

  // OptixBuildInput buildInput = {
  //   .type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES,
  //   .triangleArray = buildInputTriangleArray
  // };

  // OptixAccelBufferSizes accelBufferSizes = {};
  // optixAccelComputeMemoryUsage(context, &accelBuildOptions, &buildInput, 1, &accelBufferSizes);

  // CUdeviceptr accelerationStructureBufferDevice = 0;
  // CUdeviceptr scratchBufferDevice = 0;
  // cudaMalloc((void**)&accelerationStructureBufferDevice, accelBufferSizes.outputSizeInBytes);
  // cudaMalloc((void**)&scratchBufferDevice, accelBufferSizes.tempSizeInBytes);

  // OptixTraversableHandle outputHandle = 0;
  // optixAccelBuild(
  //   context, 
  //   0, 
  //   &accelBuildOptions, 
  //   &buildInput, 
  //   1, 
  //   scratchBufferDevice, 
  //   accelBufferSizes.tempSizeInBytes, 
  //   accelerationStructureBufferDevice,
  //   accelBufferSizes.outputSizeInBytes, 
  //   &outputHandle, 
  //   NULL, 
  //   0
  // );
}