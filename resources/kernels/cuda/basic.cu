
#include <hip/hip_runtime.h>
#define FLT_MAX 10000000

enum KernelMode {
  KERNEL_MODE_LINEAR,
  KERNEL_MODE_TILE
};

struct LinearBVHNode {
  float boundsMin[3];
  float boundsMax[3];

  union {
    int primitivesOffset;
    int secondChildOffset;
  };

  unsigned short primitiveCount;
  unsigned char axis;
  unsigned char pad[1];
};

struct Primitive {
  float positionA[3];
  float positionB[3];
  float positionC[3];
  float normalA[3];
  float normalB[3];
  float normalC[3];
  int materialIndex;
};

struct Material {
  float diffuse[3];
  float ior;
  float dissolve;
  float emission[3];
};

struct LightContainer {
  unsigned int count;
  unsigned int primitives[64];
};

struct Camera {
  float position[3];
  float yaw;
  float pitch;
  float roll;
  unsigned int frameCount;
};

struct Ray {
  float4 origin;
  float4 direction;
};

struct RayPayload {
  int primitiveIndex;
  int hitType;
  float t;
  float u;
  float v;
};

inline __device__ float3 operator*(float3 a, float b) { return make_float3(a.x * b, a.y * b, a.z * b); }
inline __device__ float4 operator*(float a, float4 b) { return make_float4(a * b.x, a * b.y, a * b.z, a * b.w); }
inline __device__ float3 operator+(float3 a, float3 b) { return make_float3(a.x + b.x, a.y + b.y, a.z + b.z); }
inline __device__ float4 operator+(float4 a, float4 b) { return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w); }
inline __device__ float4 operator-(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }
inline __device__ float4 cross(float4 a, float4 b) { return make_float4(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x, a.w); }
inline __device__ float dot(float4 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w; }

inline __device__ 
float4 reflect(float4 incidentDirection, float4 normal) {
  float cosI = -dot(normal, incidentDirection);
  return incidentDirection + 2.0f * cosI * normal;
}

inline __device__
float4 refract(float4 incidentDirection, float4 normal, float firstIOR, float secondIOR) {
  float n = firstIOR / secondIOR;
  float cosI = -dot(normal, incidentDirection);
  float sinT2 = n * n * (1.0 - cosI * cosI);
  float cosT = sqrt(1.0 - sinT2);
  return n * incidentDirection + (n * cosI - cosT) * normal;
}

__device__
const float* getBounds(int dirIsNeg, const float* boundsMin, const float* boundsMax) {
  return (dirIsNeg == 0) ? boundsMin : boundsMax;
}

__device__
bool intersectTriangle(RayPayload* rayPayload, Ray ray, Primitive primitive) {
  const float EPSILON = 0.0000001;
  float4 positionA = make_float4(primitive.positionA[0], primitive.positionA[1], primitive.positionA[2], 1);
  float4 positionB = make_float4(primitive.positionB[0], primitive.positionB[1], primitive.positionB[2], 1);
  float4 positionC = make_float4(primitive.positionC[0], primitive.positionC[1], primitive.positionC[2], 1);

  float4 v0v1 = positionB - positionA;
  float4 v0v2 = positionC - positionA;
  float4 pvec = cross(ray.direction, v0v2);
  float det = dot(v0v1, pvec);

  if (fabs(det) < EPSILON) {
    return false;
  }

  float invDet = 1 / det;

  float4 tvec = ray.origin - positionA;
  float u = dot(tvec,pvec) * invDet;
  if (u < 0 || u > 1) {
    return false;
  }

  float4 qvec = cross(tvec, v0v1);
  float v = dot(ray.direction, qvec) * invDet;
  if (v < 0 || u + v > 1) {
    return false;
  }

  float t = dot(v0v2, qvec) * invDet;

  if (t < rayPayload->t) {
    rayPayload->t = t;
    rayPayload->u = u;
    rayPayload->v = v;

    return true;
  }

  return false;
}

__device__
bool intersectBounds(Ray ray, float4 invDir, int dirIsNeg[3], const float* boundsMin, const float* boundsMax) {
  float tMin = (getBounds(dirIsNeg[0], boundsMin, boundsMax)[0] - ray.origin.x) * invDir.x;
  float tMax = (getBounds(1 - dirIsNeg[0], boundsMin, boundsMax)[0] - ray.origin.x) * invDir.x;
  float tyMin = (getBounds(dirIsNeg[1], boundsMin, boundsMax)[1] - ray.origin.y) * invDir.y;
  float tyMax = (getBounds(1 - dirIsNeg[1], boundsMin, boundsMax)[1] - ray.origin.y) * invDir.y;

  if (tMin > tyMax || tyMin > tMax) return false;
  if (tyMin > tMin) tMin = tyMin;
  if (tyMax < tMax) tMax = tyMax;

  float tzMin = (getBounds(dirIsNeg[2], boundsMin, boundsMax)[2] - ray.origin.z) * invDir.z;
  float tzMax = (getBounds(1 - dirIsNeg[2], boundsMin, boundsMax)[2] - ray.origin.z) * invDir.z;

  if (tMin > tzMax || tzMin > tMax) return false;
  if (tzMin > tMin) tMin = tzMin;
  if (tzMax < tMax) tMax = tzMax;
  return (tMax > 0);
}

__device__
void intersect(RayPayload* rayPayload, Ray ray, LinearBVHNode* nodes, Primitive* primitives) {
  float4 invDir = make_float4(1.0 / ray.direction.x, 1.0 / ray.direction.y, 1.0 / ray.direction.z, 0);
  int dirIsNeg[3] = {invDir.x < 0, invDir.y < 0, invDir.z < 0};

  int toVisitOffset = 0, currentNodeIndex = 0;
  int nodesToVisit[64];
  while (true) {
    const LinearBVHNode* node = &nodes[currentNodeIndex];

    if (intersectBounds(ray, invDir, dirIsNeg, node->boundsMin, node->boundsMax)) {
      if (node->primitiveCount > 0) {
        for (int i = 0; i < node->primitiveCount; i++) {
          if (intersectTriangle(rayPayload, ray, primitives[node->primitivesOffset])) {
            rayPayload->primitiveIndex = node->primitivesOffset;
            rayPayload->hitType = 1;
          }
        }
        if (toVisitOffset == 0) {
          break;
        }
        currentNodeIndex = nodesToVisit[--toVisitOffset];
      }
      else {
        if (dirIsNeg[node->axis]) {
          nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
          currentNodeIndex = node->secondChildOffset;
        } else {
          nodesToVisit[toVisitOffset++] = node->secondChildOffset;
          currentNodeIndex = currentNodeIndex + 1;
        }
      }
    }
    else {
      if (toVisitOffset == 0) {
        break;
      }
      currentNodeIndex = nodesToVisit[--toVisitOffset];
    }
  }
}

__device__
void intersectIgnorePrimitiveIndex(RayPayload* rayPayload, 
                                   Ray ray, 
                                   LinearBVHNode* nodes, 
                                   Primitive* primitives, 
                                   int ignorePrimitiveIndex) {

  float4 invDir = make_float4(1.0 / ray.direction.x, 1.0 / ray.direction.y, 1.0 / ray.direction.z, 0);
  int dirIsNeg[3] = {invDir.x < 0, invDir.y < 0, invDir.z < 0};

  int toVisitOffset = 0, currentNodeIndex = 0;
  int nodesToVisit[64];
  while (true) {
    const LinearBVHNode* node = &nodes[currentNodeIndex];

    if (intersectBounds(ray, invDir, dirIsNeg, node->boundsMin, node->boundsMax)) {
      if (node->primitiveCount > 0) {
        for (int i = 0; i < node->primitiveCount; i++) {
          if (node->primitivesOffset != ignorePrimitiveIndex && intersectTriangle(rayPayload, ray, primitives[node->primitivesOffset])) {
            rayPayload->primitiveIndex = node->primitivesOffset;
            rayPayload->hitType = 1;
          }
        }
        if (toVisitOffset == 0) {
          break;
        }
        currentNodeIndex = nodesToVisit[--toVisitOffset];
      }
      else {
        if (dirIsNeg[node->axis]) {
          nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
          currentNodeIndex = node->secondChildOffset;
        } else {
          nodesToVisit[toVisitOffset++] = node->secondChildOffset;
          currentNodeIndex = currentNodeIndex + 1;
        }
      }
    }
    else {
      if (toVisitOffset == 0) {
        break;
      }
      currentNodeIndex = nodesToVisit[--toVisitOffset];
    }
  }
}

__device__
void traceRayThroughLens(LinearBVHNode* linearNodes, 
                         Primitive* primitives, 
                         Material* materials,
                         RayPayload* rayPayload,
                         Ray* ray) {

  float3 outputColor = make_float3(0, 0, 0);

  Primitive* primitive = &primitives[rayPayload->primitiveIndex];
  Material* material = &materials[primitive->materialIndex];

  float3 barycentrics = make_float3(1.0 - rayPayload->u - rayPayload->v, rayPayload->u, rayPayload->v);
  float3 positionA = make_float3(primitive->positionA[0], primitive->positionA[1], primitive->positionA[2]);
  float3 positionB = make_float3(primitive->positionB[0], primitive->positionB[1], primitive->positionB[2]);
  float3 positionC = make_float3(primitive->positionC[0], primitive->positionC[1], primitive->positionC[2]);
  float3 position = positionA * barycentrics.x + positionB * barycentrics.y + positionC * barycentrics.z;
  float3 normalA = make_float3(primitive->normalA[0], primitive->normalA[1], primitive->normalA[2]);
  float3 normalB = make_float3(primitive->normalB[0], primitive->normalB[1], primitive->normalB[2]);
  float3 normalC = make_float3(primitive->normalC[0], primitive->normalC[1], primitive->normalC[2]);
  float3 normal = normalA * barycentrics.x + normalB * barycentrics.y + normalC * barycentrics.z;

  float4 transmissionDirection = refract(ray->direction, make_float4(normal.x, normal.y, normal.z, 0.0), 1.0, material->ior);

  RayPayload rayPayload2 = {0, 0, FLT_MAX, 0, 0};
  Ray ray2 = {make_float4(position.x, position.y, position.z, 1.0), transmissionDirection};
  intersectIgnorePrimitiveIndex(&rayPayload2, ray2, linearNodes, primitives, rayPayload->primitiveIndex);

  primitive = &primitives[rayPayload2.primitiveIndex];
  material = &materials[primitive->materialIndex];

  barycentrics = make_float3(1.0 - rayPayload2.u - rayPayload2.v, rayPayload2.u, rayPayload2.v);
  positionA = make_float3(primitive->positionA[0], primitive->positionA[1], primitive->positionA[2]);
  positionB = make_float3(primitive->positionB[0], primitive->positionB[1], primitive->positionB[2]);
  positionC = make_float3(primitive->positionC[0], primitive->positionC[1], primitive->positionC[2]);
  position = positionA * barycentrics.x + positionB * barycentrics.y + positionC * barycentrics.z;
  normalA = make_float3(primitive->normalA[0], primitive->normalA[1], primitive->normalA[2]);
  normalB = make_float3(primitive->normalB[0], primitive->normalB[1], primitive->normalB[2]);
  normalC = make_float3(primitive->normalC[0], primitive->normalC[1], primitive->normalC[2]);
  normal = normalA * barycentrics.x + normalB * barycentrics.y + normalC * barycentrics.z;

  transmissionDirection = refract(transmissionDirection, make_float4(-normal.x, -normal.y, -normal.z, 0.0), material->ior, 1.0);

  rayPayload->primitiveIndex = 0;
  rayPayload->hitType = 0;
  rayPayload->t = FLT_MAX;
  rayPayload->u = 0;
  rayPayload->v = 0;

  ray->origin = make_float4(position.x, position.y, position.z, 1.0);
  ray->direction = transmissionDirection;

  intersectIgnorePrimitiveIndex(rayPayload, *ray, linearNodes, primitives, rayPayload2.primitiveIndex);
}

__device__
float3 shade(LinearBVHNode* linearNodes, 
             Primitive* primitives, 
             Material* materials,
             Ray cameraRay) {

  float3 outputColor = make_float3(0, 0, 0);

  RayPayload rayPayload = {0, 0, FLT_MAX, 0, 0};
  Ray ray = cameraRay;
  intersect(&rayPayload, ray, linearNodes, primitives);

  if (rayPayload.hitType == 1) {
    Primitive* primitive = &primitives[rayPayload.primitiveIndex];
    Material* material = &materials[primitive->materialIndex];

    if (material->dissolve < 1.0) {
      traceRayThroughLens(linearNodes, primitives, materials, &rayPayload, &ray);

      if (rayPayload.hitType == 1) {
        primitive = &primitives[rayPayload.primitiveIndex];
        material = &materials[primitive->materialIndex];
      }
    }

    outputColor = make_float3(material->diffuse[0], material->diffuse[1], material->diffuse[2]);
  }

  return outputColor;
}

extern "C" __global__
void linearKernel(LinearBVHNode* linearNodes, 
                  Primitive* primitives,
                  Material* materials,
                  LightContainer* lightContainers,
                  Camera* camera,
                  float* output, 
                  int width, 
                  int height, 
                  int depth) {

  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  float4 cameraPosition = make_float4(camera->position[0], camera->position[1], camera->position[2], 1);
  float4 filmPosition = make_float4(((float)idx / width) - 0.5f, ((float)idy / height) - 0.5f, 0, 1);
  float4 aperaturePosition = make_float4(0.0, 0.0, 5, 1);

  Ray ray = {cameraPosition + filmPosition, aperaturePosition - filmPosition};
  float newX = (cos(camera->yaw) * ray.direction.x) + (sin(camera->yaw) * ray.direction.z);
  float newZ = (-sin(camera->yaw) * ray.direction.x) + (cos(camera->yaw) * ray.direction.z);
  ray.direction.x = newX;
  ray.direction.z = newZ;
  float3 outputColor = shade(linearNodes, primitives, materials, ray);

  output[id + 0] = outputColor.x;
  output[id + 1] = outputColor.y;
  output[id + 2] = outputColor.z;
}

extern "C" __global__
void tileKernel(LinearBVHNode* linearNodes, 
                Primitive* primitives,
                Material* materials,
                LightContainer* lightContainers,
                Camera* camera,
                float* output, 
                int width, 
                int height, 
                int depth) {

  int currentBlock = blockIdx.y * gridDim.x + blockIdx.x;
  int idy = ((currentBlock / gridDim.x) * blockDim.y) + threadIdx.y;
  int idx = ((currentBlock % gridDim.x) * blockDim.x) + threadIdx.x;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  float4 cameraPosition = make_float4(camera->position[0], camera->position[1], camera->position[2], 1);
  float4 filmPosition = make_float4(((float)idx / width) - 0.5f, ((float)idy / height) - 0.5f, 0, 1);
  float4 aperaturePosition = make_float4(0.0, 0.0, 5, 1);

  Ray ray = {cameraPosition + filmPosition, aperaturePosition - filmPosition};
  float newX = (cos(camera->yaw) * ray.direction.x) + (sin(camera->yaw) * ray.direction.z);
  float newZ = (-sin(camera->yaw) * ray.direction.x) + (cos(camera->yaw) * ray.direction.z);
  ray.direction.x = newX;
  ray.direction.z = newZ;
  float3 outputColor = shade(linearNodes, primitives, materials, ray);

  output[id + 0] = outputColor.x;
  output[id + 1] = outputColor.y;
  output[id + 2] = outputColor.z;
}