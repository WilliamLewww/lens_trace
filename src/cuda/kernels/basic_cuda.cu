#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include "lens_trace/structures.h"

#define FLT_MAX 10000000

struct LinearBVHNode {
  float boundsMin[3];
  float boundsMax[3];

  union {
    int primitivesOffset;
    int secondChildOffset;
  };

  ushort primitiveCount;
  unsigned char axis;
  unsigned char pad[1];
};

struct Primitive {
  float positionA[3];
  float positionB[3];
  float positionC[3];
  float normalA[3];
  float normalB[3];
  float normalC[3];
  int materialIndex;
};

struct Material {
  float diffuse[3];
  float ior;
  float dissolve;
  float emission[3];
};

struct LightContainer {
  uint32_t count;
  uint32_t primitives[64];
};

struct Camera {
  float position[3];
  float yaw;
  float pitch;
  float roll;
  uint32_t frameCount;
};

struct Ray {
  float4 origin;
  float4 direction;
};

struct RayPayload {
  int primitiveIndex;
  int hitType;
  float t;
  float u;
  float v;
};

inline __device__ float3 operator*(float3 a, float b) { return make_float3(a.x * b, a.y * b, a.z * b); }
inline __device__ float4 operator*(float a, float4 b) { return make_float4(a * b.x, a * b.y, a * b.z, a * b.w); }
inline __device__ float3 operator+(float3 a, float3 b) { return make_float3(a.x + b.x, a.y + b.y, a.z + b.z); }
inline __device__ float4 operator+(float4 a, float4 b) { return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w); }
inline __device__ float4 operator-(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }
inline __device__ float4 cross(float4 a, float4 b) { return make_float4(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x, a.w); }
inline __device__ float dot(float4 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w; }

inline __device__ 
float4 reflect(float4 incidentDirection, float4 normal) {
  float cosI = -dot(normal, incidentDirection);
  return incidentDirection + 2.0f * cosI * normal;
}

inline __device__
float4 refract(float4 incidentDirection, float4 normal, float firstIOR, float secondIOR) {
  float n = firstIOR / secondIOR;
  float cosI = -dot(normal, incidentDirection);
  float sinT2 = n * n * (1.0 - cosI * cosI);
  float cosT = sqrt(1.0 - sinT2);
  return n * incidentDirection + (n * cosI - cosT) * normal;
}

__device__
const float* getBounds(int dirIsNeg, const float* boundsMin, const float* boundsMax) {
  return (dirIsNeg == 0) ? boundsMin : boundsMax;
}

__device__
bool intersectTriangle(RayPayload* rayPayload, Ray ray, Primitive primitive) {
  const float EPSILON = 0.0000001;
  float4 positionA = make_float4(primitive.positionA[0], primitive.positionA[1], primitive.positionA[2], 1);
  float4 positionB = make_float4(primitive.positionB[0], primitive.positionB[1], primitive.positionB[2], 1);
  float4 positionC = make_float4(primitive.positionC[0], primitive.positionC[1], primitive.positionC[2], 1);

  float4 v0v1 = positionB - positionA;
  float4 v0v2 = positionC - positionA;
  float4 pvec = cross(ray.direction, v0v2);
  float det = dot(v0v1, pvec);

  if (fabs(det) < EPSILON) {
    return false;
  }

  float invDet = 1 / det;

  float4 tvec = ray.origin - positionA;
  float u = dot(tvec,pvec) * invDet;
  if (u < 0 || u > 1) {
    return false;
  }

  float4 qvec = cross(tvec, v0v1);
  float v = dot(ray.direction, qvec) * invDet;
  if (v < 0 || u + v > 1) {
    return false;
  }

  float t = dot(v0v2, qvec) * invDet;

  if (t < rayPayload->t) {
    rayPayload->t = t;
    rayPayload->u = u;
    rayPayload->v = v;

    return true;
  }

  return false;
}

__device__
bool intersectBounds(Ray ray, float4 invDir, int dirIsNeg[3], const float boundsMin[3], const float boundsMax[3]) {
  float tMin = (getBounds(dirIsNeg[0], boundsMin, boundsMax)[0] - ray.origin.x) * invDir.x;
  float tMax = (getBounds(1 - dirIsNeg[0], boundsMin, boundsMax)[0] - ray.origin.x) * invDir.x;
  float tyMin = (getBounds(dirIsNeg[1], boundsMin, boundsMax)[1] - ray.origin.y) * invDir.y;
  float tyMax = (getBounds(1 - dirIsNeg[1], boundsMin, boundsMax)[1] - ray.origin.y) * invDir.y;

  if (tMin > tyMax || tyMin > tMax) return false;
  if (tyMin > tMin) tMin = tyMin;
  if (tyMax < tMax) tMax = tyMax;

  float tzMin = (getBounds(dirIsNeg[2], boundsMin, boundsMax)[2] - ray.origin.z) * invDir.z;
  float tzMax = (getBounds(1 - dirIsNeg[2], boundsMin, boundsMax)[2] - ray.origin.z) * invDir.z;

  if (tMin > tzMax || tzMin > tMax) return false;
  if (tzMin > tMin) tMin = tzMin;
  if (tzMax < tMax) tMax = tzMax;
  return (tMax > 0);
}

__device__
void intersect(RayPayload* rayPayload, Ray ray, LinearBVHNode* nodes, Primitive* primitives) {
  float4 invDir = make_float4(1.0 / ray.direction.x, 1.0 / ray.direction.y, 1.0 / ray.direction.z, 0);
  int dirIsNeg[3] = {invDir.x < 0, invDir.y < 0, invDir.z < 0};

  int toVisitOffset = 0, currentNodeIndex = 0;
  int nodesToVisit[64];
  while (true) {
    const LinearBVHNode* node = &nodes[currentNodeIndex];

    if (intersectBounds(ray, invDir, dirIsNeg, node->boundsMin, node->boundsMax)) {
      if (node->primitiveCount > 0) {
        for (int i = 0; i < node->primitiveCount; i++) {
          if (intersectTriangle(rayPayload, ray, primitives[node->primitivesOffset])) {
            rayPayload->primitiveIndex = node->primitivesOffset;
            rayPayload->hitType = 1;
          }
        }
        if (toVisitOffset == 0) {
          break;
        }
        currentNodeIndex = nodesToVisit[--toVisitOffset];
      }
      else {
        if (dirIsNeg[node->axis]) {
          nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
          currentNodeIndex = node->secondChildOffset;
        } else {
          nodesToVisit[toVisitOffset++] = node->secondChildOffset;
          currentNodeIndex = currentNodeIndex + 1;
        }
      }
    }
    else {
      if (toVisitOffset == 0) {
        break;
      }
      currentNodeIndex = nodesToVisit[--toVisitOffset];
    }
  }
}

__device__
void intersectIgnorePrimitiveIndex(RayPayload* rayPayload, 
                                   Ray ray, 
                                   LinearBVHNode* nodes, 
                                   Primitive* primitives, 
                                   int ignorePrimitiveIndex) {

  float4 invDir = make_float4(1.0 / ray.direction.x, 1.0 / ray.direction.y, 1.0 / ray.direction.z, 0);
  int dirIsNeg[3] = {invDir.x < 0, invDir.y < 0, invDir.z < 0};

  int toVisitOffset = 0, currentNodeIndex = 0;
  int nodesToVisit[64];
  while (true) {
    const LinearBVHNode* node = &nodes[currentNodeIndex];

    if (intersectBounds(ray, invDir, dirIsNeg, node->boundsMin, node->boundsMax)) {
      if (node->primitiveCount > 0) {
        for (int i = 0; i < node->primitiveCount; i++) {
          if (node->primitivesOffset != ignorePrimitiveIndex && intersectTriangle(rayPayload, ray, primitives[node->primitivesOffset])) {
            rayPayload->primitiveIndex = node->primitivesOffset;
            rayPayload->hitType = 1;
          }
        }
        if (toVisitOffset == 0) {
          break;
        }
        currentNodeIndex = nodesToVisit[--toVisitOffset];
      }
      else {
        if (dirIsNeg[node->axis]) {
          nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
          currentNodeIndex = node->secondChildOffset;
        } else {
          nodesToVisit[toVisitOffset++] = node->secondChildOffset;
          currentNodeIndex = currentNodeIndex + 1;
        }
      }
    }
    else {
      if (toVisitOffset == 0) {
        break;
      }
      currentNodeIndex = nodesToVisit[--toVisitOffset];
    }
  }
}

__device__
void traceRayThroughLens(LinearBVHNode* linearNodes, 
                         Primitive* primitives, 
                         Material* materials,
                         RayPayload* rayPayload,
                         Ray* ray) {

  float3 outputColor = make_float3(0, 0, 0);

  Primitive* primitive = &primitives[rayPayload->primitiveIndex];
  Material* material = &materials[primitive->materialIndex];

  float3 barycentrics = make_float3(1.0 - rayPayload->u - rayPayload->v, rayPayload->u, rayPayload->v);
  float3 positionA = make_float3(primitive->positionA[0], primitive->positionA[1], primitive->positionA[2]);
  float3 positionB = make_float3(primitive->positionB[0], primitive->positionB[1], primitive->positionB[2]);
  float3 positionC = make_float3(primitive->positionC[0], primitive->positionC[1], primitive->positionC[2]);
  float3 position = positionA * barycentrics.x + positionB * barycentrics.y + positionC * barycentrics.z;
  float3 normalA = make_float3(primitive->normalA[0], primitive->normalA[1], primitive->normalA[2]);
  float3 normalB = make_float3(primitive->normalB[0], primitive->normalB[1], primitive->normalB[2]);
  float3 normalC = make_float3(primitive->normalC[0], primitive->normalC[1], primitive->normalC[2]);
  float3 normal = normalA * barycentrics.x + normalB * barycentrics.y + normalC * barycentrics.z;

  float4 transmissionDirection = refract(ray->direction, make_float4(normal.x, normal.y, normal.z, 0.0), 1.0, material->ior);

  RayPayload rayPayload2 = {0, 0, FLT_MAX, 0, 0};
  Ray ray2 = {make_float4(position.x, position.y, position.z, 1.0), transmissionDirection};
  intersectIgnorePrimitiveIndex(&rayPayload2, ray2, linearNodes, primitives, rayPayload->primitiveIndex);

  primitive = &primitives[rayPayload2.primitiveIndex];
  material = &materials[primitive->materialIndex];

  barycentrics = make_float3(1.0 - rayPayload2.u - rayPayload2.v, rayPayload2.u, rayPayload2.v);
  positionA = make_float3(primitive->positionA[0], primitive->positionA[1], primitive->positionA[2]);
  positionB = make_float3(primitive->positionB[0], primitive->positionB[1], primitive->positionB[2]);
  positionC = make_float3(primitive->positionC[0], primitive->positionC[1], primitive->positionC[2]);
  position = positionA * barycentrics.x + positionB * barycentrics.y + positionC * barycentrics.z;
  normalA = make_float3(primitive->normalA[0], primitive->normalA[1], primitive->normalA[2]);
  normalB = make_float3(primitive->normalB[0], primitive->normalB[1], primitive->normalB[2]);
  normalC = make_float3(primitive->normalC[0], primitive->normalC[1], primitive->normalC[2]);
  normal = normalA * barycentrics.x + normalB * barycentrics.y + normalC * barycentrics.z;

  transmissionDirection = refract(transmissionDirection, make_float4(-normal.x, -normal.y, -normal.z, 0.0), material->ior, 1.0);

  rayPayload->primitiveIndex = 0;
  rayPayload->hitType = 0;
  rayPayload->t = FLT_MAX;
  rayPayload->u = 0;
  rayPayload->v = 0;

  ray->origin = make_float4(position.x, position.y, position.z, 1.0);
  ray->direction = transmissionDirection;

  intersectIgnorePrimitiveIndex(rayPayload, *ray, linearNodes, primitives, rayPayload2.primitiveIndex);
}

__device__
float3 shade(LinearBVHNode* linearNodes, 
             Primitive* primitives, 
             Material* materials,
             Ray cameraRay) {

  float3 outputColor = make_float3(0, 0, 0);

  RayPayload rayPayload = {0, 0, FLT_MAX, 0, 0};
  Ray ray = cameraRay;
  intersect(&rayPayload, ray, linearNodes, primitives);

  if (rayPayload.hitType == 1) {
    Primitive* primitive = &primitives[rayPayload.primitiveIndex];
    Material* material = &materials[primitive->materialIndex];

    if (material->dissolve < 1.0) {
      traceRayThroughLens(linearNodes, primitives, materials, &rayPayload, &ray);

      if (rayPayload.hitType == 1) {
        primitive = &primitives[rayPayload.primitiveIndex];
        material = &materials[primitive->materialIndex];
      }
    }

    outputColor = make_float3(material->diffuse[0], material->diffuse[1], material->diffuse[2]);
  }

  return outputColor;
}

__global__
void linearKernel(LinearBVHNode* linearNodes, 
                  Primitive* primitives,
                  Material* materials,
                  LightContainer* lightContainers,
                  Camera* camera,
                  float* output, 
                  int width, 
                  int height, 
                  int depth) {

  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  float4 cameraPosition = make_float4(camera->position[0], camera->position[1], camera->position[2], 1);
  float4 filmPosition = make_float4(((float)idx / width) - 0.5f, ((float)idy / height) - 0.5f, 0, 1);
  float4 aperaturePosition = make_float4(0.0, 0.0, 5, 1);

  Ray ray = {cameraPosition + filmPosition, aperaturePosition - filmPosition};
  float newX = (cos(camera->yaw) * ray.direction.x) + (sin(camera->yaw) * ray.direction.z);
  float newZ = (-sin(camera->yaw) * ray.direction.x) + (cos(camera->yaw) * ray.direction.z);
  ray.direction.x = newX;
  ray.direction.z = newZ;
  float3 outputColor = shade(linearNodes, primitives, materials, ray);

  output[id + 0] = outputColor.x;
  output[id + 1] = outputColor.y;
  output[id + 2] = outputColor.z;
}

__global__
void tileKernel(LinearBVHNode* linearNodes, 
                Primitive* primitives,
                Material* materials,
                LightContainer* lightContainers,
                Camera* camera,
                float* output, 
                int width, 
                int height, 
                int depth) {

  int currentBlock = blockIdx.y * gridDim.x + blockIdx.x;
  int idy = ((currentBlock / gridDim.x) * blockDim.y) + threadIdx.y;
  int idx = ((currentBlock % gridDim.x) * blockDim.x) + threadIdx.x;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  float4 cameraPosition = make_float4(camera->position[0], camera->position[1], camera->position[2], 1);
  float4 filmPosition = make_float4(((float)idx / width) - 0.5f, ((float)idy / height) - 0.5f, 0, 1);
  float4 aperaturePosition = make_float4(0.0, 0.0, 5, 1);

  Ray ray = {cameraPosition + filmPosition, aperaturePosition - filmPosition};
  float newX = (cos(camera->yaw) * ray.direction.x) + (sin(camera->yaw) * ray.direction.z);
  float newZ = (-sin(camera->yaw) * ray.direction.x) + (cos(camera->yaw) * ray.direction.z);
  ray.direction.x = newX;
  ray.direction.z = newZ;
  float3 outputColor = shade(linearNodes, primitives, materials, ray);

  output[id + 0] = outputColor.x;
  output[id + 1] = outputColor.y;
  output[id + 2] = outputColor.z;
}

extern "C" void basic_cuda_kernelWrappers(void* linearNodeBuffer,
                                          uint64_t linearNodeBufferSize,
                                          void* primitiveBuffer,
                                          uint64_t primitiveBufferSize,
                                          void* materialBuffer,
                                          uint64_t materialBufferSize,
                                          void* lightContainerBuffer,
                                          uint64_t lightContainerBufferSize,
                                          void* cameraBuffer,
                                          uint64_t cameraBufferSize,
                                          void* outputBuffer, 
                                          uint64_t imageDimensions[3],
                                          uint64_t blockSize[2],
                                          KernelMode kernelMode) {

  dim3 block(blockSize[0], blockSize[1]);
  dim3 grid((imageDimensions[0] + block.x - 1) / block.x, (imageDimensions[1] + block.y - 1) / block.y);

  void* linearNodeBufferDevice;
  hipMalloc(&linearNodeBufferDevice, linearNodeBufferSize);
  hipMemcpy(linearNodeBufferDevice, linearNodeBuffer, linearNodeBufferSize, hipMemcpyHostToDevice);

  void* primitiveBufferDevice;
  hipMalloc(&primitiveBufferDevice, primitiveBufferSize);
  hipMemcpy(primitiveBufferDevice, primitiveBuffer, primitiveBufferSize, hipMemcpyHostToDevice);

  void* materialBufferDevice;
  hipMalloc(&materialBufferDevice, materialBufferSize);
  hipMemcpy(materialBufferDevice, materialBuffer, materialBufferSize, hipMemcpyHostToDevice);

  void* lightContainerBufferDevice;
  hipMalloc(&lightContainerBufferDevice, lightContainerBufferSize);
  hipMemcpy(lightContainerBufferDevice, lightContainerBuffer, lightContainerBufferSize, hipMemcpyHostToDevice);

  void* cameraBufferDevice;
  hipMalloc(&cameraBufferDevice, cameraBufferSize);
  hipMemcpy(cameraBufferDevice, cameraBuffer, cameraBufferSize, hipMemcpyHostToDevice);

  void* outputBufferDevice;
  hipMalloc(&outputBufferDevice, sizeof(float) * imageDimensions[0] * imageDimensions[1] * imageDimensions[2]);

  if (kernelMode == KERNEL_MODE_LINEAR) {
    linearKernel<<<grid, block>>>(
      (LinearBVHNode*)linearNodeBufferDevice, 
      (Primitive*)primitiveBufferDevice, 
      (Material*)materialBufferDevice,
      (LightContainer*)lightContainerBufferDevice,
      (Camera*)cameraBufferDevice, 
      (float*)outputBufferDevice, 
      imageDimensions[0], 
      imageDimensions[1], 
      imageDimensions[2]
    );
  }

  if (kernelMode == KERNEL_MODE_TILE) {
    tileKernel<<<grid, block>>>(
      (LinearBVHNode*)linearNodeBufferDevice, 
      (Primitive*)primitiveBufferDevice, 
      (Material*)materialBufferDevice,
      (LightContainer*)lightContainerBufferDevice,
      (Camera*)cameraBufferDevice, 
      (float*)outputBufferDevice, 
      imageDimensions[0], 
      imageDimensions[1], 
      imageDimensions[2]
    );
  }
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s\n", hipGetErrorString(error));
  }

  hipMemcpy(outputBuffer, outputBufferDevice, sizeof(float) * imageDimensions[0] * imageDimensions[1] * imageDimensions[2], hipMemcpyDeviceToHost);
  hipFree(outputBufferDevice);
  hipFree(cameraBufferDevice);
  hipFree(lightContainerBufferDevice);
  hipFree(materialBufferDevice);
  hipFree(primitiveBufferDevice);
  hipFree(linearNodeBufferDevice);
}