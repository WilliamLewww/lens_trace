
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

struct LinearBVHNode {
  float boundsMin[3];
  float boundsMax[3];

  union {
    int primitivesOffset;
    int secondChildOffset;
  };

  ushort primitiveCount;
  unsigned char axis;
  unsigned char pad[1];
};

struct Primitive {
  float positionA[3];
  float positionB[3];
  float positionC[3];
  float normalA[3];
  float normalB[3];
  float normalC[3];
  int materialIndex;
};

struct Material {
  float diffuse[3];
  float ior;
  float dissolve;
};

struct Camera {
  float position[3];
  float yaw;
};

__global__
void linearKernel(LinearBVHNode* linearNodes, 
                  Primitive* primitives,
                  Material* materials, 
                  Camera* camera,
                  float* output, 
                  int width, 
                  int height, 
                  int depth) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  output[id] = id;
}

extern "C" void linearKernelWrapper(void* linearNodeBuffer,
                                    uint64_t linearNodeBufferSize,
                                    void* primitiveBuffer,
                                    uint64_t primitiveBufferSize,
                                    void* materialBuffer,
                                    uint64_t materialBufferSize,
                                    void* cameraBuffer,
                                    uint64_t cameraBufferSize,
                                    void* outputBuffer, 
                                    int width, 
                                    int height, 
                                    int depth) {

  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  void* linearNodeBufferDevice;
  hipMalloc(&linearNodeBufferDevice, linearNodeBufferSize);
  hipMemcpy(linearNodeBufferDevice, linearNodeBuffer, linearNodeBufferSize, hipMemcpyHostToDevice);

  void* primitiveBufferDevice;
  hipMalloc(&primitiveBufferDevice, primitiveBufferSize);
  hipMemcpy(primitiveBufferDevice, primitiveBuffer, primitiveBufferSize, hipMemcpyHostToDevice);

  void* materialBufferDevice;
  hipMalloc(&materialBufferDevice, materialBufferSize);
  hipMemcpy(materialBufferDevice, materialBuffer, materialBufferSize, hipMemcpyHostToDevice);

  void* cameraBufferDevice;
  hipMalloc(&cameraBufferDevice, cameraBufferSize);
  hipMemcpy(cameraBufferDevice, cameraBuffer, cameraBufferSize, hipMemcpyHostToDevice);

  void* outputBufferDevice;
  hipMalloc(&outputBufferDevice, sizeof(float) * width * height * depth);

  linearKernel<<<grid, block>>>(
    (LinearBVHNode*)linearNodeBufferDevice, 
    (Primitive*)primitiveBufferDevice, 
    (Material*)materialBufferDevice, 
    (Camera*)cameraBufferDevice, 
    (float*)outputBufferDevice, 
    width, 
    height, 
    depth
  );
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s\n", hipGetErrorString(error));
  }

  hipMemcpy(outputBuffer, outputBufferDevice, sizeof(float) * width * height * depth, hipMemcpyDeviceToHost);
  hipFree(outputBufferDevice);
}

extern "C" void tileKernelWrapper() {

}