
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define FLT_MAX 10000000

struct LinearBVHNode {
  float boundsMin[3];
  float boundsMax[3];

  union {
    int primitivesOffset;
    int secondChildOffset;
  };

  ushort primitiveCount;
  unsigned char axis;
  unsigned char pad[1];
};

struct Primitive {
  float positionA[3];
  float positionB[3];
  float positionC[3];
  float normalA[3];
  float normalB[3];
  float normalC[3];
  int materialIndex;
};

struct Material {
  float diffuse[3];
  float ior;
  float dissolve;
};

struct Camera {
  float position[3];
  float yaw;
};

struct Ray {
  float4 origin;
  float4 direction;
};

struct RayPayload {
  int primitiveIndex;
  int hitType;
  float t;
  float u;
  float v;
};

inline __device__ float4 operator+(float4 a, float4 b) { return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w); }
inline __device__ float4 operator-(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }
inline __device__ float4 cross(float4 a, float4 b) { return make_float4(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x, a.w); }
inline __device__ float dot(float4 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w; }

__device__
const float* getBounds(int dirIsNeg, const float* boundsMin, const float* boundsMax) {
  return (dirIsNeg == 0) ? boundsMin : boundsMax;
}

__device__
bool intersectTriangle(RayPayload* rayPayload, Ray ray, Primitive primitive) {
  const float EPSILON = 0.0000001;
  float4 positionA = make_float4(primitive.positionA[0], primitive.positionA[1], primitive.positionA[2], 1);
  float4 positionB = make_float4(primitive.positionB[0], primitive.positionB[1], primitive.positionB[2], 1);
  float4 positionC = make_float4(primitive.positionC[0], primitive.positionC[1], primitive.positionC[2], 1);

  float4 v0v1 = positionB - positionA;
  float4 v0v2 = positionC - positionA;
  float4 pvec = cross(ray.direction, v0v2);
  float det = dot(v0v1, pvec);

  if (fabs(det) < EPSILON) {
    return false;
  }

  float invDet = 1 / det;

  float4 tvec = ray.origin - positionA;
  float u = dot(tvec,pvec) * invDet;
  if (u < 0 || u > 1) {
    return false;
  }

  float4 qvec = cross(tvec, v0v1);
  float v = dot(ray.direction, qvec) * invDet;
  if (v < 0 || u + v > 1) {
    return false;
  }

  float t = dot(v0v2, qvec) * invDet;

  if (t < rayPayload->t) {
    rayPayload->t = t;
    rayPayload->u = u;
    rayPayload->v = v;

    return true;
  }

  return false;
}

__device__
bool intersectBounds(Ray ray, float4 invDir, int dirIsNeg[3], const float boundsMin[3], const float boundsMax[3]) {
  float tMin = (getBounds(dirIsNeg[0], boundsMin, boundsMax)[0] - ray.origin.x) * invDir.x;
  float tMax = (getBounds(1 - dirIsNeg[0], boundsMin, boundsMax)[0] - ray.origin.x) * invDir.x;
  float tyMin = (getBounds(dirIsNeg[1], boundsMin, boundsMax)[1] - ray.origin.y) * invDir.y;
  float tyMax = (getBounds(1 - dirIsNeg[1], boundsMin, boundsMax)[1] - ray.origin.y) * invDir.y;

  if (tMin > tyMax || tyMin > tMax) return false;
  if (tyMin > tMin) tMin = tyMin;
  if (tyMax < tMax) tMax = tyMax;

  float tzMin = (getBounds(dirIsNeg[2], boundsMin, boundsMax)[2] - ray.origin.z) * invDir.z;
  float tzMax = (getBounds(1 - dirIsNeg[2], boundsMin, boundsMax)[2] - ray.origin.z) * invDir.z;

  if (tMin > tzMax || tzMin > tMax) return false;
  if (tzMin > tMin) tMin = tzMin;
  if (tzMax < tMax) tMax = tzMax;
  return (tMax > 0);
}

__device__
void intersect(RayPayload* rayPayload, Ray ray, LinearBVHNode* nodes, Primitive* primitives) {
  float4 invDir = make_float4(1.0 / ray.direction.x, 1.0 / ray.direction.y, 1.0 / ray.direction.z, 0);
  int dirIsNeg[3] = {invDir.x < 0, invDir.y < 0, invDir.z < 0};

  int toVisitOffset = 0, currentNodeIndex = 0;
  int nodesToVisit[64];
  while (true) {
    const LinearBVHNode* node = &nodes[currentNodeIndex];

    if (intersectBounds(ray, invDir, dirIsNeg, node->boundsMin, node->boundsMax)) {
      if (node->primitiveCount > 0) {
        for (int i = 0; i < node->primitiveCount; i++) {
          if (intersectTriangle(rayPayload, ray, primitives[node->primitivesOffset])) {
            rayPayload->primitiveIndex = node->primitivesOffset;
            rayPayload->hitType = 1;
          }
        }
        if (toVisitOffset == 0) {
          break;
        }
        currentNodeIndex = nodesToVisit[--toVisitOffset];
      }
      else {
        if (dirIsNeg[node->axis]) {
          nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
          currentNodeIndex = node->secondChildOffset;
        } else {
          nodesToVisit[toVisitOffset++] = node->secondChildOffset;
          currentNodeIndex = currentNodeIndex + 1;
        }
      }
    }
    else {
      if (toVisitOffset == 0) {
        break;
      }
      currentNodeIndex = nodesToVisit[--toVisitOffset];
    }
  }
}

__device__
float3 shade(LinearBVHNode* linearNodes, 
             Primitive* primitives, 
             Material* materials,
             Ray cameraRay) {

  float3 outputColor = make_float3(0, 0, 0);

  RayPayload rayPayload = {0, 0, FLT_MAX, 0, 0};
  Ray ray = cameraRay;
  intersect(&rayPayload, ray, linearNodes, primitives);

  if (rayPayload.hitType == 1) {
    Primitive* primitive = &primitives[rayPayload.primitiveIndex];
    Material* material = &materials[primitive->materialIndex];

    outputColor = make_float3(material->diffuse[0], material->diffuse[1], material->diffuse[2]);
  }

  return outputColor;
}

__global__
void linearKernel(LinearBVHNode* linearNodes, 
                  Primitive* primitives,
                  Material* materials, 
                  Camera* camera,
                  float* output, 
                  int width, 
                  int height, 
                  int depth) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  float4 cameraPosition = make_float4(camera->position[0], camera->position[1], camera->position[2], 1);
  float4 filmPosition = make_float4(((float)idx / width) - 0.5f, ((float)idy / height) - 0.5f, 0, 1);
  float4 aperaturePosition = make_float4(0.0, 0.0, 5, 1);

  Ray ray = {cameraPosition + filmPosition, aperaturePosition - filmPosition};
  ray.direction.x = (cos(camera->yaw) * ray.direction.x) + (sin(camera->yaw) * ray.direction.z);
  ray.direction.z = (-sin(camera->yaw) * ray.direction.x) + (cos(camera->yaw) * ray.direction.z);
  float3 outputColor = shade(linearNodes, primitives, materials, ray);

  output[id + 0] = outputColor.x;
  output[id + 1] = outputColor.y;
  output[id + 2] = outputColor.z;
}

extern "C" void linearKernelWrapper(void* linearNodeBuffer,
                                    uint64_t linearNodeBufferSize,
                                    void* primitiveBuffer,
                                    uint64_t primitiveBufferSize,
                                    void* materialBuffer,
                                    uint64_t materialBufferSize,
                                    void* cameraBuffer,
                                    uint64_t cameraBufferSize,
                                    void* outputBuffer, 
                                    int width, 
                                    int height, 
                                    int depth) {

  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  void* linearNodeBufferDevice;
  hipMalloc(&linearNodeBufferDevice, linearNodeBufferSize);
  hipMemcpy(linearNodeBufferDevice, linearNodeBuffer, linearNodeBufferSize, hipMemcpyHostToDevice);

  void* primitiveBufferDevice;
  hipMalloc(&primitiveBufferDevice, primitiveBufferSize);
  hipMemcpy(primitiveBufferDevice, primitiveBuffer, primitiveBufferSize, hipMemcpyHostToDevice);

  void* materialBufferDevice;
  hipMalloc(&materialBufferDevice, materialBufferSize);
  hipMemcpy(materialBufferDevice, materialBuffer, materialBufferSize, hipMemcpyHostToDevice);

  void* cameraBufferDevice;
  hipMalloc(&cameraBufferDevice, cameraBufferSize);
  hipMemcpy(cameraBufferDevice, cameraBuffer, cameraBufferSize, hipMemcpyHostToDevice);

  void* outputBufferDevice;
  hipMalloc(&outputBufferDevice, sizeof(float) * width * height * depth);

  linearKernel<<<grid, block>>>(
    (LinearBVHNode*)linearNodeBufferDevice, 
    (Primitive*)primitiveBufferDevice, 
    (Material*)materialBufferDevice, 
    (Camera*)cameraBufferDevice, 
    (float*)outputBufferDevice, 
    width, 
    height, 
    depth
  );
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s\n", hipGetErrorString(error));
  }

  hipMemcpy(outputBuffer, outputBufferDevice, sizeof(float) * width * height * depth, hipMemcpyDeviceToHost);
  hipFree(outputBufferDevice);
}

extern "C" void tileKernelWrapper() {

}