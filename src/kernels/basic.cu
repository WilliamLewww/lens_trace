
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void linearKernel(float* pOutputBuffer, int width, int height, int depth) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int id = (idy * width + idx) * depth;

  if (idx >= width || idy >= height) {
    return;
  }

  pOutputBuffer[id] = id;
}

extern "C" void linearKernelWrapper(float* pOutputBuffer, int width, int height, int depth) {
  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  float* pOutputBufferDevice;
  hipMalloc(&pOutputBufferDevice, sizeof(float) * width * height * depth);

  linearKernel<<<grid, block>>>(pOutputBufferDevice, width, height, depth);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s\n", hipGetErrorString(error));
  }

  hipMemcpy(pOutputBuffer, pOutputBufferDevice, sizeof(float) * width * height * depth, hipMemcpyDeviceToHost);
  hipFree(pOutputBufferDevice);
}

extern "C" void tileKernelWrapper() {

}