
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void linearKernel(float* pOutputBuffer, int width, int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  int id = idy * width + idx;

  if (idx >= width || idy >= height) {
    return;
  }

  pOutputBuffer[id] = id;
}

extern "C" void linearKernelWrapper(float* pOutputBuffer, int width, int height) {
  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  float* pOutputBufferDevice;
  hipMalloc(&pOutputBufferDevice, sizeof(float) * width * height);

  linearKernel<<<grid, block>>>(pOutputBufferDevice, width, height);
  hipDeviceSynchronize();

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("%s\n", hipGetErrorString(error));
  }

  hipMemcpy(pOutputBuffer, pOutputBufferDevice, sizeof(float) * width * height, hipMemcpyDeviceToHost);
  hipFree(pOutputBufferDevice);
}

extern "C" void tileKernelWrapper() {

}